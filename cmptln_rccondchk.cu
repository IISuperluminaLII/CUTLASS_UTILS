#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2025 NVIDIA CORPORATION.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Runtime probe that mirrors the SM120 backward kernel's TMEM->register staging path and records the
 * linear TMEM indices each compute-lane touches.  Duplicate indices in the output highlight the
 * race-condition that `cute::copy` flags during compilation.
 **************************************************************************************************/

#include <hip/hip_runtime.h>

#include <iostream>
#include <map>
#include <string>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cute/atom/copy_traits.hpp"
#include "cute/algorithm/tuple_algorithms.hpp"

#include "sm120/prefill/dense/collective/fmha_fusion.hpp"
#include "sm120/prefill/dense/sm120_kernel_traits.hpp"
#include "sm120/prefill/dense/kernel/sm120_fmha_bwd_kernel_tma_warpspecialized.hpp"

namespace sm120_index_probe {

using KernelTraits = flash::Sm120WorkstationConfig;
using ProblemShape = cute::tuple<int, int, int, int, cute::tuple<int, int>>;
using Element = cutlass::bfloat16_t;
using ElementAcc = float;
using TileShape = typename KernelTraits::TileShapeFmhaBwd;
using Mask = cutlass::fmha::collective::ResidualMask;

using Kernel = cutlass::fmha::kernel::Sm120FmhaBwdKernelTmaWarpSpecialized<
    KernelTraits, ProblemShape, Element, ElementAcc, TileShape, Mask>;

constexpr int kComputeThreads =
    Kernel::kNumReduceWarps * cutlass::NumThreadsPerWarp;
constexpr int kMaxElementsPerThread = 1024;  // generous upper bound

#define CUDA_CHECK(cmd)                                                             \
  do {                                                                              \
    hipError_t status = (cmd);                                                     \
    if (status != hipSuccess) {                                                    \
      std::cerr << "CUDA error: " << hipGetErrorString(status) << " (" << __LINE__ \
                << ")\n";                                                           \
      std::exit(EXIT_FAILURE);                                                      \
    }                                                                               \
  } while (0)

__global__ void probe_dq_indices(int* src, int* dst, int stride, int* counts,
                                 int* coord0, int* coord1, int* coord2) {
  using namespace cute;

  auto frag_builder =
      partition_fragment_C(typename Kernel::TiledMmaDSK{},
                           select<0, 1>(typename Kernel::TileShapeDSK{}));
  auto tDQtDQ = frag_builder(make_coord(_, _), _0{}, _0{});
  tDQtDQ.data() = Kernel::TmemAllocation::kDQ;

  auto cDQ = make_identity_tensor(take<0, 2>(typename Kernel::TileShapeDSK{}));

  int t = threadIdx.x;
  int lane = t % cutlass::NumThreadsPerWarp;
  if (t >= kComputeThreads) {
    return;
  }

  using CopyAtom = cute::Copy_Atom<cute::UniversalCopy<uint128_t>, ElementAcc>;
  auto copy_op = make_cotiled_copy(CopyAtom{}, tDQtDQ.layout(), tDQtDQ.layout());
  auto thread_copy = copy_op.get_slice(t);

  auto tTR_tDQ = thread_copy.partition_S(tDQtDQ);
  auto tTR_cDQ = thread_copy.partition_D(cDQ);
  auto tTR_rDQ = make_tensor<ElementAcc>(shape(tTR_cDQ));
  constexpr int kRank = decltype(rank(tTR_cDQ))::value;
  static_assert(kRank >= 1, "DQ tensor rank must be >= 1 for this probe.");

  int elements = static_cast<int>(size(tTR_tDQ));
  counts[t] = elements;

  int warp = t / cutlass::NumThreadsPerWarp;
  int per_warp = elements / Kernel::kNumReduceWarps;
  for (int i = warp * per_warp + lane;
       i < (warp + 1) * per_warp && i < stride;
       i += cutlass::NumThreadsPerWarp) {
    auto src_coord = tTR_tDQ.get_1d_coord(i);
    auto dst_coord = tTR_rDQ.get_1d_coord(i);

    int src_offset = static_cast<int>(tTR_tDQ.layout()(src_coord));
    int dst_offset = static_cast<int>(tTR_rDQ.layout()(dst_coord));

    int coord_vals[3] = {-1, -1, -1};
    int coord_idx = 0;
    cute::for_each(dst_coord, [&](auto const& val) {
      if (coord_idx < 3) {
        coord_vals[coord_idx] = static_cast<int>(val);
      }
      ++coord_idx;
    });
    coord0[t * stride + i] = coord_vals[0];
    coord1[t * stride + i] = coord_vals[1];
    coord2[t * stride + i] = coord_vals[2];

    src[t * stride + i] = src_offset;
    dst[t * stride + i] = dst_offset;
  }
}

struct CollisionReport {
  std::map<int, std::vector<std::string>> src_hits;
  std::map<int, std::vector<std::string>> dst_hits;
  int total_elements = 0;
  bool truncated = false;
};

CollisionReport analyse(const std::vector<int>& src,
                        const std::vector<int>& dst,
                        const std::vector<int>& counts,
                        const std::vector<int>& coord0,
                        const std::vector<int>& coord1,
                        const std::vector<int>& coord2,
                        int stride) {
  CollisionReport report{};

  for (int thread = 0; thread < kComputeThreads; ++thread) {
    int elements = counts[thread];
    if (elements < 0) {
      continue;
    }
    if (elements > stride) {
      report.truncated = true;
      elements = stride;
    }
    report.total_elements += elements;

    int warp = thread / cutlass::NumThreadsPerWarp;
    int lane = thread % cutlass::NumThreadsPerWarp;

    for (int i = 0; i < elements; ++i) {
      int src_offset = src[thread * stride + i];
      int dst_offset = dst[thread * stride + i];
      std::string info = "warp=" + std::to_string(warp) + "/lane=" +
                         std::to_string(lane) + "/elem=" + std::to_string(i) +
                         "/dst=" + std::to_string(dst_offset) + "/coord=" +
                         "(" + std::to_string(coord0[thread * stride + i]) +
                         "," + std::to_string(coord1[thread * stride + i]) +
                         "," + std::to_string(coord2[thread * stride + i]) +
                         ")";

      report.src_hits[src_offset].push_back(info);
      report.dst_hits[dst_offset].push_back(info);
    }
  }

  return report;
}

template <class Map>
bool dump_collisions(const char* label, const Map& hits, int max_examples = 8) {
  bool ok = true;
  for (auto const& kv : hits) {
    if (kv.second.size() < 2) {
      continue;
    }
    if (ok) {
      std::cout << "---- " << label << " collisions ----\n";
    }
    ok = false;
    std::cout << "index " << kv.first << " hit " << kv.second.size()
              << " times\n";
    int printed = 0;
    for (auto const& info : kv.second) {
      if (printed++ >= max_examples) {
        std::cout << "  ... +" << (kv.second.size() - max_examples)
                  << " more\n";
        break;
      }
      std::cout << "  " << info << "\n";
    }
  }
  return ok;
}

}  // namespace sm120_index_probe

int main() {
  using namespace sm120_index_probe;

  std::vector<int> host_src(kComputeThreads * kMaxElementsPerThread, -1);
  std::vector<int> host_dst(kComputeThreads * kMaxElementsPerThread, -1);
  std::vector<int> host_counts(kComputeThreads, -1);
  std::vector<int> host_coord0(kComputeThreads * kMaxElementsPerThread, -1);
  std::vector<int> host_coord1(kComputeThreads * kMaxElementsPerThread, -1);
  std::vector<int> host_coord2(kComputeThreads * kMaxElementsPerThread, -1);

  int* dev_src = nullptr;
  int* dev_dst = nullptr;
  int* dev_counts = nullptr;
  int* dev_coord0 = nullptr;
  int* dev_coord1 = nullptr;
  int* dev_coord2 = nullptr;

  CUDA_CHECK(hipMalloc(&dev_src,
                        host_src.size() * sizeof(int)));
  CUDA_CHECK(hipMalloc(&dev_dst,
                        host_dst.size() * sizeof(int)));
  CUDA_CHECK(hipMalloc(&dev_counts,
                        host_counts.size() * sizeof(int)));
  CUDA_CHECK(hipMalloc(&dev_coord0,
                        host_coord0.size() * sizeof(int)));
  CUDA_CHECK(hipMalloc(&dev_coord1,
                        host_coord1.size() * sizeof(int)));
  CUDA_CHECK(hipMalloc(&dev_coord2,
                        host_coord2.size() * sizeof(int)));

  CUDA_CHECK(hipMemset(dev_src, 0xFF, host_src.size() * sizeof(int)));
  CUDA_CHECK(hipMemset(dev_dst, 0xFF, host_dst.size() * sizeof(int)));
  CUDA_CHECK(hipMemset(dev_counts, 0xFF, host_counts.size() * sizeof(int)));
  CUDA_CHECK(hipMemset(dev_coord0, 0xFF, host_coord0.size() * sizeof(int)));
  CUDA_CHECK(hipMemset(dev_coord1, 0xFF, host_coord1.size() * sizeof(int)));
  CUDA_CHECK(hipMemset(dev_coord2, 0xFF, host_coord2.size() * sizeof(int)));

  probe_dq_indices<<<1, kComputeThreads>>>(dev_src, dev_dst, kMaxElementsPerThread,
                               dev_counts, dev_coord0, dev_coord1, dev_coord2);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(host_src.data(), dev_src,
                        host_src.size() * sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(host_dst.data(), dev_dst,
                        host_dst.size() * sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(host_counts.data(), dev_counts,
                        host_counts.size() * sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(host_coord0.data(), dev_coord0,
                        host_coord0.size() * sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(host_coord1.data(), dev_coord1,
                        host_coord1.size() * sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(host_coord2.data(), dev_coord2,
                        host_coord2.size() * sizeof(int),
                        hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(dev_src));
  CUDA_CHECK(hipFree(dev_dst));
  CUDA_CHECK(hipFree(dev_counts));
  CUDA_CHECK(hipFree(dev_coord0));
  CUDA_CHECK(hipFree(dev_coord1));
  CUDA_CHECK(hipFree(dev_coord2));

  auto report = analyse(host_src, host_dst, host_counts,
                        host_coord0, host_coord1, host_coord2,
                        kMaxElementsPerThread);

  if (report.truncated) {
    std::cout << "Warning: some per-thread element lists were truncated. "
                 "Increase kMaxElementsPerThread for full coverage.\n";
  }

  std::cout << "Total recorded elements: " << report.total_elements
            << ", unique TMEM slots: " << report.src_hits.size()
            << ", unique register slots: " << report.dst_hits.size()
            << "\n";

  bool src_ok = dump_collisions("TMEM", report.src_hits);
  bool dst_ok = dump_collisions("register", report.dst_hits);

  if (src_ok && dst_ok) {
    std::cout << "No index collisions detected.\n";
    return 0;
  }

  std::cout << "Index collisions detected. Inspect the collision log for the "
               "responsible warp/lane pairs.\n";
  return 1;
}
template <class Coord>
std::string coord_to_string(Coord const& coord) {
  std::ostringstream oss;
  oss << "(";
  bool first = true;
  cute::for_each(coord, [&](auto const& value) {
    if (!first) {
      oss << ",";
    }
    first = false;
    oss << static_cast<int>(value);
  });
  oss << ")";
  return oss.str();
}
